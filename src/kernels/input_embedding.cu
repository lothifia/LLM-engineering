#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/input_embedding.h"
#include "src/utils/cuda_debug_utils.cuh"
template<typename T>
__global__ void embeddingFunctor(const int* input_ids,
               T* output, 
               const T* embed_table,
               const int max_context_token_num,
               const int hidden_size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // 全局idx
    while (index < max_context_token_num * hidden_size) {// 线程 数据不匹配时进行循环处理 这里保证了 index.max == output.size()
        int id = input_ids[index / hidden_size]; // 多个线程处理， 将每个H内的数值分配给H个线程来做。 通过inputs 定位到目标词在词表中的id
        output[index] = embed_table[id * hidden_size + index % hidden_size]; // 多个线程并行取出
        index += blockDim.x * gridDim.x;
    }
}
template<typename T>
__global__ void embeddingFunctor_h(const int* input_ids, T* output, const T* embed_table, const int max_context_token_num, const int hidden_size) {
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int tot_data = hidden_size * max_context_token_num;
    for(int idx = gidx; idx < tot_data; idx += stride) {
        int vacab_idx = input_ids[gidx / hidden_size];
        output[gidx] = embed_table[vacab_idx * hidden_size + gidx % hidden_size];
    }
}

template<typename T>
void launchInputEmbedding(TensorWrapper<int>* input_ids,    // INT [token num]
                          TensorWrapper<T>* output,       // FP32 [token num, hidden_size] = [token num, 4096]
                          EmbeddingWeight<T>* embed_table// FP32 [vocab_size, hidden_size]
                          ) {
    const int blockSize = 256;
    const int max_context_token_num = output->shape[0]; // token num
    const int hidden_size = output->shape[1];
    const int gridSize = 2048;
    LLM_CHECK_WITH_INFO(max_context_token_num == input_ids->shape[0], "input ids 1st shape should equal to 1st shape of output");
    embeddingFunctor_h<T><<<gridSize, blockSize>>>(input_ids->data,
                                                 output->data,
                                                 embed_table->data,
                                                 max_context_token_num,
                                                 hidden_size);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}
/*显式实例化在此代码中的作用：

明确支持的精度类型：只允许 float 和 half

确保CUDA内核代码生成：为每种精度生成特定内核

避免跨编译单元问题：保证单一定义

优化编译时间：避免重复实例化

提供清晰接口：明确库支持的类型*/
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<float>* output,       
                                   EmbeddingWeight<float>* embed_table);
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<half>* output,       
                                   EmbeddingWeight<half>* embed_table);
