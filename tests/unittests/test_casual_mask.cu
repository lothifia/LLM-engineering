#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include "src/kernels/build_casual_mask.h"
void PrintCausalMask(const float* mask, int batch_size, int max_q_len, int max_k_len) {
    for (int b = 0; b < batch_size; ++b) {
        std::cout << "Batch " << b << ":\n";
        for (int q = 0; q < max_q_len; ++q) {
            std::cout << "q=" << q << " | ";
            for (int k = 0; k < max_k_len; ++k) {
                // 线性 index = b * (max_q_len * max_k_len) + q * max_k_len + k
                int idx = b * max_q_len * max_k_len + q * max_k_len + k;
                std::cout << static_cast<int>(mask[idx]) << " ";
            }
            std::cout << "\n";
        }
        std::cout << "------------------------\n";
    }
}
// (RussWong)note: this kernel's CPU implementation is absolutely right.
// when you are implementing LLMs inference on CPU, you can reuse the CPU kernel
// we compare the kernel correctnesss by eyes and result print infos
void CPUbuildCausalMask(float* mask, 
                        const int* q_lens,  //input lens, shape=[batch size]
                        const int* k_lens,  //context lens, shape=[batch size]
                        int max_q_len, 
                        int max_k_len,
                        int batch_size) {
    for(int b = 0; b < batch_size; b++){
        int start = b * max_q_len * max_k_len;
        int q = q_lens[b];
        int k = k_lens[b];
        for(int i = 0; i < max_q_len; i++) {
            for(int j = 0; j < max_k_len; j++) {
                if(j <= i + (k - q) && i < q && j < k) {
                    mask[start + i * max_k_len + j] = 1.0f;
                } else {
                    mask[start + i * max_k_len + j] = 0.0f;   
                }
            }
        }
    }
}
bool CheckResult(float* CPUres, float* GPUres, const int size) {
    for(int i = 0; i < size; i++) {
        if(fabs(CPUres[i] - GPUres[i]) > 1e-6){
            printf("the %dth res is wrong, CPU mask = %f, GPU mask = %f\n", i, CPUres[i], GPUres[i]);
            return false;
        }
    }
    return true;
}
// (RussWong)note:
// `./causalmask` to test fp32 GPU build causal mask kernel
int main() {
    const int batch_size = 1;
    const int max_q_len = 4;
    const int max_k_len = 8;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    const int mask_size = batch_size * max_q_len * max_k_len;
    int* h_q_lens;
    int* d_q_lens;
    h_q_lens = (int*)malloc(sizeof(int) * batch_size);
    hipMalloc((void**)&d_q_lens, sizeof(int) * batch_size);
    int* h_k_lens;
    int* d_k_lens;
    h_k_lens = (int*)malloc(sizeof(int) * batch_size);
    hipMalloc((void**)&d_k_lens, sizeof(int) * batch_size);

    float* d_mask;
    float* h_mask = (float*)malloc(sizeof(float) * mask_size);
    hipMalloc((void**)&d_mask, sizeof(float) * mask_size);

    for(int i = 0; i < batch_size; i++) {
       h_q_lens[i] = 4;
    }
    for(int i = 0; i < batch_size; i++) {
       h_k_lens[i] = 8;
    }
    CHECK(hipMemcpy(d_q_lens, h_q_lens, sizeof(int) * batch_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_k_lens, h_k_lens, sizeof(int) * batch_size, hipMemcpyHostToDevice));
    DataType type_float = getTensorType<float>();
    DataType type_int = getTensorType<int>();
    TensorWrapper<float>* mask = new TensorWrapper<float>(Device::GPU, 
                                                        type_float,
                                                        {batch_size, max_q_len, max_k_len}, 
                                                        d_mask);
    TensorWrapper<int>* q_lens = new TensorWrapper<int>(Device::GPU, 
                                                        type_int,
                                                        {batch_size}, 
                                                        d_q_lens);
    TensorWrapper<int>* k_lens = new TensorWrapper<int>(Device::GPU, 
                                                        type_int,
                                                        {batch_size}, 
                                                        d_k_lens);
    launchBuildCausalMasks(mask, q_lens, k_lens);
    // debug info, better to retain: std::cout << "after launch kernel" << std::endl;
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    CHECK(hipMemcpy(h_mask, d_mask, sizeof(float) * mask_size, hipMemcpyDeviceToHost));

    float* CPUmask = (float*)malloc(sizeof(float) * mask_size);
    CPUbuildCausalMask(CPUmask, h_q_lens, h_k_lens, max_q_len, max_k_len, batch_size);
    if (CheckResult(CPUmask, h_mask, mask_size)) {
        printf("test passed!\n");
    }
    PrintCausalMask(CPUmask, batch_size, max_q_len, max_k_len);

    // debug info, better to retain: std::cout << "before free" << std::endl;
    free(h_q_lens);
    free(h_k_lens);
    free(h_mask);
    free(CPUmask);
    hipFree(d_q_lens);
    hipFree(d_k_lens);
    hipFree(d_mask);
}
