#include "hip/hip_runtime.h"
#include "src/kernels/cal_paddingoffset.h"
// shape:
    //seq_lengths:[batch size]
    //cum_seqlens:[batch size + 1],first ele is 0
    //padding_offset:[batch size * max q len]
// note: the point is to calc padding offset and cum offset
// TODO: we first use serial algo, then can enhance to CUDA scan algo

__global__ void CalPaddingoffset(int*        padding_offset, 
                                int*         cum_seqlens, // 表示累积到当前坐标的的句子长度 
                                const int*   input_lengths, //actual input lens
                                const int    batch_size,
                                const int    max_q_len) {
    int ind = 0;
    int cum_offset = 0;
    int total_seqlen = 0;
    for(int b = 0; b < batch_size; b++) { // 所有句子
        int seqlen = input_lengths[b]; // 该句长度

        cum_seqlens[b] = total_seqlen; //  当前句子之前的句子长度
        // each token in one seq has same cum offset
        for (int i = 0; i < seqlen; i++) { // 对句子内的每个token
            padding_offset[ind] = cum_offset; // paddingOffest 修改成 cum_offest 表示之前累积了多少padding 即之前的prefix
            ind++; // 在padding_offest中是个长数组.
        }
        cum_offset += max_q_len - seqlen; // prefix叠加
        total_seqlen += seqlen; // 句子总长度增加
    }
    cum_seqlens[batch_size] = total_seqlen;// 跟新全部句子                            

}

void launchCalPaddingoffset(TensorWrapper<int>* padding_offset, 
                            TensorWrapper<int>* cum_seqlens,
                            TensorWrapper<int>* input_lengths)//actual input lens
{
    const int batch_size = padding_offset->shape[0];      // 句子长度                      
    const int max_q_len = padding_offset->shape[1];       // 最大长度
    LLM_CHECK_WITH_INFO(batch_size == input_lengths->shape[0], "input lenghts numbers should equal to padding offset bs dim!") ;                        
    LLM_CHECK_WITH_INFO(batch_size == cum_seqlens->shape[0] - 1, "cum seqlen numbers should equal to padding offset bs dim + 1!") ;                        
    CalPaddingoffset<<<1, 1>>>( 
        padding_offset->data, cum_seqlens->data, input_lengths->data, batch_size, max_q_len
    );
}